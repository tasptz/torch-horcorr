#include "hip/hip_runtime.h"
#include <stdint.h>

template <typename T>
inline const T &min(const T &a, const T &b) {
    return a <= b ? a : b;
}

__global__ void kernelForward(
    const float *left,
    const float *right,
    float *output,
    uint32_t leftWidth) {

    const int row = threadIdx.x + blockIdx.x * blockDim.x;
    const int indexLeft = row * leftWidth * DIM;
    const int rightWidth = leftWidth - NWINDOWS + 1;
    const int indexRight = row * rightWidth * DIM;
    const int indexOut = row * rightWidth * NWINDOWS;
    float *pOutput = output + indexOut;

    float vecLeft[NWINDOWS][DIM];
    const float *pLeft = left + indexLeft;
    memcpy(vecLeft, pLeft, NWINDOWS * DIM * sizeof(float));
    pLeft += NWINDOWS * DIM;

    float vecRight[DIM];
    const float *pRight = right + indexRight;

    int idxRing = 0;
    for (int w = 0; w < rightWidth; ++w) {
        memcpy(vecRight, pRight, DIM * sizeof(float));
        pRight += DIM;

        if (w > 0) {
            memcpy(vecLeft[idxRing], pLeft, DIM * sizeof(float));
            idxRing = (idxRing + 1) % NWINDOWS;
            pLeft += DIM;
        }

        #pragma unroll
        for (int i = 0; i < NWINDOWS; ++i) {
            const int idx = (idxRing + i) % NWINDOWS;
            float sum = 0.f;

            #pragma unroll
            for (int j = 0; j < DIM; ++j) {
                sum += vecLeft[idx][j] * vecRight[j];
            }
            *pOutput++ = sum;
        }
    }
}

void runKernelForward(
    uint32_t numBlocks, uint32_t blockSize,
    const float *left, const float *right, float *output,
    uint32_t leftWidth) {

    kernelForward<<<numBlocks, blockSize>>>(left, right, output, leftWidth);
}

__global__ void kernelRightBackward(
    const float *left,
    const float *grad,
    float *output,
    uint32_t leftWidth) {

    const int row = threadIdx.x + blockIdx.x * blockDim.x;
    const int indexLeft = row * leftWidth * DIM;
    const float *pLeft = left + indexLeft;
    const int rightWidth = leftWidth - NWINDOWS + 1;
    const int indexGrad = row * rightWidth * NWINDOWS;
    const float *pGrad = grad + indexGrad;
    const int indexOut = row * rightWidth * DIM;
    float *pOutput = output + indexOut;

    for (int w = 0; w < rightWidth; ++w) {
        #pragma unroll
        for (int d = 0; d < DIM; ++d) {
            float s = 0.f;
            #pragma unroll
            for (int n = 0; n < NWINDOWS; ++n) {
                s += pLeft[d + n * DIM] * pGrad[n];
            }
            *pOutput++ = s;
        }
        pLeft += DIM;
        pGrad += NWINDOWS;
    }
}

void runKernelRightBackward(
    uint32_t numBlocks, uint32_t blockSize,
    const float *left, const float *grad, float *output,
    uint32_t leftWidth) {
    kernelRightBackward<<<numBlocks, blockSize>>>(left, grad, output, leftWidth);
}

__global__ void kernelLeftBackward(
    const float *right,
    const float *grad,
    float *output,
    uint32_t rightWidth) {

    const int row = threadIdx.x + blockIdx.x * blockDim.x;
    const int indexRight = row * rightWidth * DIM;
    const float *pRight = right + indexRight;
    const float *const pRightEnd = pRight + rightWidth * DIM;
    const int leftWidth = rightWidth + NWINDOWS - 1;
    const int indexGrad = row * rightWidth * NWINDOWS;
    const float *const pGradBegin = grad + indexGrad;
    const int indexOut = row * leftWidth * DIM;
    float *pOutput = output + indexOut;

    for (int w = 0; w < leftWidth; ++w) {
        #pragma unroll
        for (int d = 0; d < DIM; ++d) {
            float s = 0.f;
            #pragma unroll
            for (int n = 0; n < min(w + 1, NWINDOWS); ++n) {
                const float *p = pRight + (d - n * DIM);
                s += p < pRightEnd ? (*p * pGradBegin[(w - n) * NWINDOWS + n]) : 0.f;
            }
            *pOutput++ = s;
        }
        pRight += DIM;
    }
}

void runKernelLeftBackward(
    uint32_t numBlocks, uint32_t blockSize,
    const float *right, const float *grad, float *output,
    uint32_t rightWidth) {
    kernelLeftBackward<<<numBlocks, blockSize>>>(right, grad, output, rightWidth);
}